#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <jni.h>
#include <hipsolver.h>
#include <assert.h>
#include <iostream>
#include <raft/linalg/cublas_wrappers.h>
#include <raft/linalg/eig.cuh>
#include <raft/linalg/svd.cuh>
#include <raft/matrix/matrix.hpp>
#include <raft/matrix/math.hpp>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <rmm/exec_policy.hpp>
#include <nvtx3.hpp>

struct java_domain {
  static constexpr char const *name{"Java"};
};

namespace {
void signFlip(
  double* input, int n_rows, int n_cols, double* components, int n_cols_comp, hipStream_t stream)
{
  auto counting = thrust::make_counting_iterator(0);
  auto m        = n_rows;

  thrust::for_each(rmm::exec_policy(stream), counting, counting + n_cols, [=] __device__(int idx) {
    int d_i = idx * m;
    int end = d_i + m;

    double max    = 0.0;
    int max_index = 0;
    for (int i = d_i; i < end; i++) {
      double val = input[i];
      if (val < 0.0) { val = -val; }
      if (val > max) {
        max       = val;
        max_index = i;
      }
    }

    if (input[max_index] < 0.0) {
      for (int i = d_i; i < end; i++) {
        input[i] = -input[i];
      }
    }
  });
}

hipblasOperation_t convertToCublasOpEnum(int int_type)
{
  switch(int_type) {
    case 0: return HIPBLAS_OP_N;
    case 1: return HIPBLAS_OP_T;
    case 2: return HIPBLAS_OP_C;
    case 3: return CUBLAS_OP_CONJG;
    default:
      throw "Invalid type enum: " + std::to_string(int_type);
      break;
  }
}
} // anonymous namespace

extern "C" {

  JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_NvtxRange_push(JNIEnv *env, jclass clazz, jstring name,
                                                          jint color_bits) {
  jclass jlexception = env->FindClass("java/lang/Exception");
  try {
    // cudf::jni::native_jstring range_name(env, name);
    const char *range_name = env->GetStringUTFChars(name, 0);
    nvtx3::color range_color(static_cast<nvtx3::color::value_type>(color_bits));
    nvtx3::event_attributes attr{range_color, range_name};
    nvtxDomainRangePushEx(nvtx3::domain::get<java_domain>(), attr.get());
  } catch (const std::bad_alloc &e) {
    env->ThrowNew(jlexception, "Error nvtx push");
  }
  // CATCH_STD(env, );
}

JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_NvtxRange_pop(JNIEnv *env, jclass clazz) {
  jclass jlexception = env->FindClass("java/lang/Exception");
  try {
    nvtxDomainRangePop(nvtx3::domain::get<java_domain>());
  } catch (const std::bad_alloc &e) {
    env->ThrowNew(jlexception, "Error nvtx pop");
  }
  // CATCH_STD(env, );
}

JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_JniRAPIDSML_dspr(JNIEnv* env, jclass, jint n, jdoubleArray x,
                                                                      jdoubleArray A) {
  jclass jlexception = env->FindClass("java/lang/Exception");
  auto size_A = env->GetArrayLength(A);

  double* dev_x;
  auto cuda_error = hipMalloc((void**)&dev_x, n * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for x");
  }

  double* dev_A;
  cuda_error = hipMalloc((void**)&dev_A, size_A * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for A");
  }

  auto* host_x = env->GetDoubleArrayElements(x, nullptr);
  cuda_error = hipMemcpyAsync(dev_x, host_x, n * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying x to device");
  }

  auto* host_A = env->GetDoubleArrayElements(A, nullptr);
  cuda_error = hipMemcpyAsync(dev_A, host_A, size_A * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying A to device");
  }

  hipblasHandle_t handle;
  auto status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error creating cuBLAS handle");
  }

  double alpha = 1.0;
  status = hipblasDspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha, dev_x, 1, dev_A);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error calling hipblasDspr");
  }

  cuda_error = hipMemcpyAsync(host_A, dev_A, size_A * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying A from device");
  }

  cuda_error = hipFree(dev_x);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing x from device");
  }

  cuda_error = hipFree(dev_A);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing A from device");
  }

  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error destroying cuBLAS handle");
  }

  env->ReleaseDoubleArrayElements(x, host_x, JNI_ABORT);
  env->ReleaseDoubleArrayElements(A, host_A, 0);
}

JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_JniRAPIDSML_dgemm(JNIEnv* env, jclass, jint transa, jint transb,
                                                                        jint m, jint n, jint k, jdouble alpha,
                                                                        jdoubleArray A, jint lda, jdoubleArray B,
                                                                        jint ldb, jdouble beta, jdoubleArray C, jint ldc, jint deviceID) {
  hipSetDevice(deviceID);
  jclass jlexception = env->FindClass("java/lang/Exception");

  raft::handle_t raft_handle;
  hipStream_t stream = raft_handle.get_stream();

  auto size_A = env->GetArrayLength(A);
  auto size_B = env->GetArrayLength(B);
  auto size_C = env->GetArrayLength(C);

  double* dev_A;
  auto cuda_error = hipMalloc((void**)&dev_A, size_A * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for A");
  }

  double* dev_B;
  cuda_error = hipMalloc((void**)&dev_B, size_B * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for B");
  }

  double* dev_C;
  cuda_error = hipMalloc((void**)&dev_C, size_C * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for C");
  }

  auto* host_A = env->GetDoubleArrayElements(A, nullptr);
  cuda_error = hipMemcpyAsync(dev_A, host_A, size_A * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying A to device");
  }

  auto* host_B = env->GetDoubleArrayElements(B, nullptr);
  cuda_error = hipMemcpyAsync(dev_B, host_B, size_B * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying B to device");
  }

  hipblasHandle_t handle;
  auto status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error creating cuBLAS handle");
  }

  status = raft::linalg::cublasgemm(handle, convertToCublasOpEnum(transa), convertToCublasOpEnum(transb), m, n, k, &alpha, dev_A, lda, dev_B, ldb, &beta,
                       dev_C, ldc, stream);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error calling hipblasDgemm");
  }

  auto* host_C = env->GetDoubleArrayElements(C, nullptr);
  cuda_error = hipMemcpyAsync(host_C, dev_C, size_C * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying C from device");
  }

  cuda_error = hipFree(dev_A);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing A from device");
  }

    cuda_error = hipFree(dev_B);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing B from device");
  }

  cuda_error = hipFree(dev_C);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing C from device");
  }

  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error destroying cuBLAS handle");
  }

  env->ReleaseDoubleArrayElements(A, host_A, JNI_ABORT);
  env->ReleaseDoubleArrayElements(B, host_B, JNI_ABORT);
  env->ReleaseDoubleArrayElements(C, host_C, 0);
}

JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_JniRAPIDSML_dgemm_1b(JNIEnv* env, jclass, jint rows_a, jint cols_b, jint cols_a,
                                                                       jdoubleArray A, jdoubleArray B, jdoubleArray C, jint deviceID) {

  hipSetDevice(deviceID);
  jclass jlexception = env->FindClass("java/lang/Exception");
  auto size_A = env->GetArrayLength(A);
  auto size_B = env->GetArrayLength(B);
  auto size_C = env->GetArrayLength(C);

  double* dev_A;
  auto cuda_error = hipMalloc((void**)&dev_A, size_A * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for A");
  }

   double* dev_B;
   cuda_error = hipMalloc((void**)&dev_B, size_B * sizeof(double));
   if (cuda_error != hipSuccess) {
     env->ThrowNew(jlexception, "Error allocating device memory for A");
   }

  double* dev_C;
  cuda_error = hipMalloc((void**)&dev_C, size_C * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for C");
  }

  auto* host_A = env->GetDoubleArrayElements(A, nullptr);
  cuda_error = hipMemcpyAsync(dev_A, host_A, size_A * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying A to device");
  }

  auto* host_B = env->GetDoubleArrayElements(B, nullptr);
  cuda_error = hipMemcpyAsync(dev_B, host_B, size_B * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying A to device");
  }

  hipblasHandle_t handle;
  auto status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error creating cuBLAS handle");
  }

  double alpha = 1.0;
  double beta = 0.0;
  status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows_a, cols_b, cols_a, &alpha, dev_A, cols_a, dev_B, cols_a, &beta,
                       dev_C, rows_a);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error calling hipblasDgemm");
  }

  auto* host_C = env->GetDoubleArrayElements(C, nullptr);
  cuda_error = hipMemcpyAsync(host_C, dev_C, size_C * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying C from device");
  }

  cuda_error = hipFree(dev_A);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing A from device");
  }

  cuda_error = hipFree(dev_C);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing C from device");
  }

  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error destroying cuBLAS handle");
  }

  env->ReleaseDoubleArrayElements(A, host_A, JNI_ABORT);
  env->ReleaseDoubleArrayElements(C, host_C, 0);
}

JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_JniRAPIDSML_calSVD
  (JNIEnv * env, jclass, jint m, jdoubleArray A, jdoubleArray U, jdoubleArray S, jint deviceID) {
    hipSetDevice(deviceID);
    raft::handle_t handle;
    hipStream_t stream = handle.get_stream();

    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;

    double *d_A = NULL;
    double *d_S = NULL;
    double *d_U = NULL;

    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double)*m*m);
    cudaStat2 = hipMalloc ((void**)&d_S  , sizeof(double)*m);
    cudaStat3 = hipMalloc ((void**)&d_U  , sizeof(double)*m*m);

    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    auto size_A = env->GetArrayLength(A);
    jdouble* host_A = env->GetDoubleArrayElements(A, JNI_FALSE);

    cudaStat1 = hipMemcpy(d_A, host_A, sizeof(double)*m*m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    auto* host_U = env->GetDoubleArrayElements(U, nullptr);
    auto cuda_error = hipMemcpyAsync(host_U, d_U, m * m * sizeof(double), hipMemcpyDefault);
    assert(hipSuccess == cuda_error);

    auto* host_S = env->GetDoubleArrayElements(S, nullptr);
    cuda_error = hipMemcpyAsync(host_S, d_S, m * sizeof(double), hipMemcpyDefault);
    assert(hipSuccess == cuda_error);

    raft::linalg::eigDC(handle, d_A, m, m, d_U, d_S, stream);
    raft::matrix::colReverse(d_U, m, m, stream);
    raft::matrix::rowReverse(d_S, m, 1, stream);
    raft::matrix::seqRoot(d_S, d_S, 1.0, m, stream, true);

    signFlip(d_U, m, m, d_U, m, stream);

    cudaStat1 = hipMemcpy(host_U , d_U , sizeof(double)*m*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(host_S , d_S , sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    if (d_A    ) hipFree(d_A);
    if (d_S    ) hipFree(d_S);
    if (d_U    ) hipFree(d_U);
    env->ReleaseDoubleArrayElements(A, host_A, JNI_ABORT);
    env->ReleaseDoubleArrayElements(U, host_U, 0);
    env->ReleaseDoubleArrayElements(S, host_S, 0);
  }

}  // extern "C"



