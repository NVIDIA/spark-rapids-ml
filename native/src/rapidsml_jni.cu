#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/sequence.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/scalar/scalar.hpp>

#include <raft/linalg/cublas_wrappers.h>
#include <raft/linalg/eig.cuh>
#include <raft/matrix/matrix.hpp>
#include <raft/matrix/math.hpp>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <jni.h>

#include "rapidsml_jni.hpp"

void signFlip(
  double* input, int n_rows, int n_cols, double* components, int n_cols_comp, hipStream_t stream) {
  auto counting = thrust::make_counting_iterator(0);
  auto m        = n_rows;

  thrust::for_each(rmm::exec_policy(stream), counting, counting + n_cols, [=] __device__(int idx) {
    int d_i = idx * m;
    int end = d_i + m;

    double max    = 0.0;
    int max_index = 0;
    for (int i = d_i; i < end; i++) {
      double val = input[i];
      if (val < 0.0) { val = -val; }
      if (val > max) {
        max       = val;
        max_index = i;
      }
    }

    if (input[max_index] < 0.0) {
      for (int i = d_i; i < end; i++) {
        input[i] = -input[i];
      }
    }
  });
}

hipblasOperation_t convertToCublasOpEnum(int int_type) {
  switch(int_type) {
    case 0: return HIPBLAS_OP_N;
    case 1: return HIPBLAS_OP_T;
    case 2: return HIPBLAS_OP_C;
    case 3: return CUBLAS_OP_CONJG;
    default:
      throw "Invalid type enum: " + std::to_string(int_type);
      break;
  }
}

long dgemm(int transa, int transb, int m, int n,int k, double alpha, double* A, int size_A, int lda,
           long B, int ldb, double beta, int ldc, int deviceID) {
    hipSetDevice(deviceID);
    raft::handle_t raft_handle;
    hipStream_t stream = raft_handle.get_stream();
    auto const *B_cv_ptr = reinterpret_cast<cudf::lists_column_view const *>(B);
    auto const child_column_view = B_cv_ptr->child();
    // init cuda stream view from rmm
    auto c_stream = rmm::cuda_stream_view(stream);

    rmm::device_buffer dev_buff_A = rmm::device_buffer(A, size_A * sizeof(double), c_stream);

    auto size_C = m * n;
    //create child column that will own the computation result
    auto child_column = cudf::make_numeric_column(cudf::data_type{cudf::type_id::FLOAT64}, size_C);
    auto child_mutable_view = child_column->mutable_view();
    auto status = raft::linalg::cublasgemm(raft_handle.get_cublas_handle(),
                                           convertToCublasOpEnum(transa),
                                           convertToCublasOpEnum(transb),
                                           m, n, k, &alpha, (double const *)dev_buff_A.data(), lda,
                                           child_column_view.data<double>(),ldb, &beta,
                                           child_mutable_view.data<double>(), ldc, stream);
    // create offset column
    auto zero = cudf::numeric_scalar<int32_t>(0, true, c_stream);
    auto step = cudf::numeric_scalar<int32_t>(m, true, c_stream);
    auto offset_column = cudf::sequence(n + 1, zero, step,
                                        rmm::mr::get_current_device_resource());

    auto target_column = cudf::make_lists_column(n, std::move(offset_column),
                                                 std::move(child_column), 0, rmm::device_buffer());

    return reinterpret_cast<long>(target_column.release());
}


extern "C" {

JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_JniRAPIDSML_dgemm(JNIEnv* env, jclass,
                                                                         jint transa, jint transb,
                                                                         jint m, jint n, jint k,
                                                                         jdouble alpha,
                                                                         jdoubleArray A, jint lda,
                                                                         jdoubleArray B,jint ldb,
                                                                         jdouble beta, jdoubleArray C,
                                                                         jint ldc, jint deviceID) {
  hipSetDevice(deviceID);
  jclass jlexception = env->FindClass("java/lang/Exception");

  raft::handle_t raft_handle;
  hipStream_t stream = raft_handle.get_stream();

  auto size_A = env->GetArrayLength(A);
  auto size_B = env->GetArrayLength(B);
  auto size_C = env->GetArrayLength(C);

  double* dev_A;
  auto cuda_error = hipMalloc((void**)&dev_A, size_A * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for A");
  }

  double* dev_B;
  cuda_error = hipMalloc((void**)&dev_B, size_B * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for B");
  }

  double* dev_C;
  cuda_error = hipMalloc((void**)&dev_C, size_C * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for C");
  }

  auto* host_A = env->GetDoubleArrayElements(A, nullptr);
  cuda_error = hipMemcpyAsync(dev_A, host_A, size_A * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying A to device");
  }

  auto* host_B = env->GetDoubleArrayElements(B, nullptr);
  cuda_error = hipMemcpyAsync(dev_B, host_B, size_B * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying B to device");
  }

  auto status = raft::linalg::cublasgemm(raft_handle.get_cublas_handle(),
                                         convertToCublasOpEnum(transa),
                                         convertToCublasOpEnum(transb), m, n, k, &alpha, dev_A, lda,
                                         dev_B, ldb, &beta, dev_C, ldc, stream);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error calling hipblasDgemm");
  }

  auto* host_C = env->GetDoubleArrayElements(C, nullptr);
  cuda_error = hipMemcpyAsync(host_C, dev_C, size_C * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying C from device");
  }

  cuda_error = hipFree(dev_A);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing A from device");
  }

    cuda_error = hipFree(dev_B);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing B from device");
  }

  cuda_error = hipFree(dev_C);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing C from device");
  }

  env->ReleaseDoubleArrayElements(A, host_A, JNI_ABORT);
  env->ReleaseDoubleArrayElements(B, host_B, JNI_ABORT);
  env->ReleaseDoubleArrayElements(C, host_C, 0);
}


JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_JniRAPIDSML_calSVD
  (JNIEnv * env, jclass, jint m, jdoubleArray A, jdoubleArray U, jdoubleArray S, jint deviceID) {
    hipSetDevice(deviceID);
    raft::handle_t handle;
    hipStream_t stream = handle.get_stream();

    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;

    double *d_A = NULL;
    double *d_S = NULL;
    double *d_U = NULL;

    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double)*m*m);
    cudaStat2 = hipMalloc ((void**)&d_S  , sizeof(double)*m);
    cudaStat3 = hipMalloc ((void**)&d_U  , sizeof(double)*m*m);

    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    auto size_A = env->GetArrayLength(A);
    jdouble* host_A = env->GetDoubleArrayElements(A, JNI_FALSE);

    cudaStat1 = hipMemcpy(d_A, host_A, sizeof(double)*m*m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    auto* host_U = env->GetDoubleArrayElements(U, nullptr);
    auto cuda_error = hipMemcpyAsync(host_U, d_U, m * m * sizeof(double), hipMemcpyDefault);
    assert(hipSuccess == cuda_error);

    auto* host_S = env->GetDoubleArrayElements(S, nullptr);
    cuda_error = hipMemcpyAsync(host_S, d_S, m * sizeof(double), hipMemcpyDefault);
    assert(hipSuccess == cuda_error);

    raft::linalg::eigDC(handle, d_A, m, m, d_U, d_S, stream);
    raft::matrix::colReverse(d_U, m, m, stream);
    raft::matrix::rowReverse(d_S, m, 1, stream);
    raft::matrix::seqRoot(d_S, d_S, 1.0, m, stream, true);

    signFlip(d_U, m, m, d_U, m, stream);

    cudaStat1 = hipMemcpy(host_U , d_U , sizeof(double)*m*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(host_S , d_S , sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    if (d_A    ) hipFree(d_A);
    if (d_S    ) hipFree(d_S);
    if (d_U    ) hipFree(d_U);
    env->ReleaseDoubleArrayElements(A, host_A, JNI_ABORT);
    env->ReleaseDoubleArrayElements(U, host_U, 0);
    env->ReleaseDoubleArrayElements(S, host_S, 0);
  }

}// extern "C"
