#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <raft/linalg/cublas_wrappers.h>
#include <raft/linalg/eig.cuh>
#include <raft/matrix/matrix.cuh>
#include <raft/matrix/math.cuh>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <rmm/exec_policy.hpp>
#include <jni.h>
#include "ml_utils.cu"

extern "C" {

JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_JniRAPIDSML_dgemm(JNIEnv* env, jclass, jint transa, jint transb,
                                                                        jint m, jint n, jint k, jdouble alpha,
                                                                        jdoubleArray A, jint lda, jdoubleArray B,
                                                                        jint ldb, jdouble beta, jdoubleArray C, jint ldc, jint deviceID) {
  hipSetDevice(deviceID);
  jclass jlexception = env->FindClass("java/lang/Exception");

  raft::handle_t raft_handle;
  hipStream_t stream = raft_handle.get_stream();

  auto size_A = env->GetArrayLength(A);
  auto size_B = env->GetArrayLength(B);
  auto size_C = env->GetArrayLength(C);

  double* dev_A;
  auto cuda_error = hipMalloc((void**)&dev_A, size_A * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for A");
  }

  double* dev_B;
  cuda_error = hipMalloc((void**)&dev_B, size_B * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for B");
  }

  double* dev_C;
  cuda_error = hipMalloc((void**)&dev_C, size_C * sizeof(double));
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error allocating device memory for C");
  }

  auto* host_A = env->GetDoubleArrayElements(A, nullptr);
  cuda_error = hipMemcpyAsync(dev_A, host_A, size_A * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying A to device");
  }

  auto* host_B = env->GetDoubleArrayElements(B, nullptr);
  cuda_error = hipMemcpyAsync(dev_B, host_B, size_B * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying B to device");
  }

  auto status = raft::linalg::cublasgemm(raft_handle.get_cublas_handle(), convertToCublasOpEnum(transa), convertToCublasOpEnum(transb), m, n, k, &alpha, dev_A, lda, dev_B, ldb, &beta,
                       dev_C, ldc, stream);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    env->ThrowNew(jlexception, "Error calling hipblasDgemm");
  }

  auto* host_C = env->GetDoubleArrayElements(C, nullptr);
  cuda_error = hipMemcpyAsync(host_C, dev_C, size_C * sizeof(double), hipMemcpyDefault);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error copying C from device");
  }

  cuda_error = hipFree(dev_A);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing A from device");
  }

    cuda_error = hipFree(dev_B);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing B from device");
  }

  cuda_error = hipFree(dev_C);
  if (cuda_error != hipSuccess) {
    env->ThrowNew(jlexception, "Error freeing C from device");
  }

  env->ReleaseDoubleArrayElements(A, host_A, JNI_ABORT);
  env->ReleaseDoubleArrayElements(B, host_B, JNI_ABORT);
  env->ReleaseDoubleArrayElements(C, host_C, 0);
}


JNIEXPORT void JNICALL Java_com_nvidia_spark_ml_linalg_JniRAPIDSML_calSVD
  (JNIEnv * env, jclass, jint m, jdoubleArray A, jdoubleArray U, jdoubleArray S, jint deviceID) {
    hipSetDevice(deviceID);
    raft::handle_t handle;
    hipStream_t stream = handle.get_stream();

    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;

    double *d_A = NULL;
    double *d_S = NULL;
    double *d_U = NULL;

    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double)*m*m);
    cudaStat2 = hipMalloc ((void**)&d_S  , sizeof(double)*m);
    cudaStat3 = hipMalloc ((void**)&d_U  , sizeof(double)*m*m);

    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    auto size_A = env->GetArrayLength(A);
    jdouble* host_A = env->GetDoubleArrayElements(A, JNI_FALSE);

    cudaStat1 = hipMemcpy(d_A, host_A, sizeof(double)*m*m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    auto* host_U = env->GetDoubleArrayElements(U, nullptr);
    auto cuda_error = hipMemcpyAsync(host_U, d_U, m * m * sizeof(double), hipMemcpyDefault);
    assert(hipSuccess == cuda_error);

    auto* host_S = env->GetDoubleArrayElements(S, nullptr);
    cuda_error = hipMemcpyAsync(host_S, d_S, m * sizeof(double), hipMemcpyDefault);
    assert(hipSuccess == cuda_error);

    raft::linalg::eigDC(handle, d_A, m, m, d_U, d_S, stream);
    raft::matrix::colReverse(d_U, m, m, stream);
    raft::matrix::rowReverse(d_S, m, 1, stream);
    raft::matrix::seqRoot(d_S, d_S, 1.0, m, stream, true);

    signFlip(d_U, m, m, d_U, m, stream);

    cudaStat1 = hipMemcpy(host_U , d_U , sizeof(double)*m*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(host_S , d_S , sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    if (d_A    ) hipFree(d_A);
    if (d_S    ) hipFree(d_S);
    if (d_U    ) hipFree(d_U);
    env->ReleaseDoubleArrayElements(A, host_A, JNI_ABORT);
    env->ReleaseDoubleArrayElements(U, host_U, 0);
    env->ReleaseDoubleArrayElements(S, host_S, 0);
  }

}// extern "C"